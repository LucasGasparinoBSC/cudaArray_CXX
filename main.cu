#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <cstdio>
#include "Array.hpp"
#include "ArrayDevice.cuh"

int main()
{
    // Create a host array of float type
    Array<float> a_h(100);

    // Create a device array
    ArrayDevice<float> a_d(100);

    // Create a chunk of a_d
    ArrayDevice<float> a_d_chunk(10);
    a_d.chunk(0,9,a_d_chunk);
    return 0;
}